#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.h"

// A dummy kernel that prints block and thread indices
__global__ void dummyKernel() {
    printf("Block %d, Thread %d\n", blockIdx.x, threadIdx.x);
}