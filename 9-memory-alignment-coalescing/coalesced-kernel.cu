#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel with coalesced memory access
__global__ void coalescedKernel(const float *input, float *output, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        output[idx] = input[idx] * 2.0f;
    }
}

int main() {
    int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(float);
    hipError_t err;

    // Allocate host memory
    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    // Initialize host array
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    // Allocate device memory
    float *d_input, *d_output;
    err = hipMalloc((void**)&d_input, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
    err = hipMalloc((void**)&d_output, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }

    // Copy input data to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Configure kernel execution
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Benchmark coalesced kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    coalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print result
    printf("Coalesced Kernel Time: %f ms\n", milliseconds);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
