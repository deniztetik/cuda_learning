#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel with non-coalesced memory access
__global__ void nonCoalescedKernel(const float *input, float *output, int N, int stride) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int index = (idx * stride) % N; // Wrap around to match workload
        output[index] = input[index] * 2.0f;
    }
}

int main() {
    int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(float);
    hipError_t err;

    // Allocate host memory
    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    // Initialize host array
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    // Allocate device memory
    float *d_input, *d_output;
    err = hipMalloc((void**)&d_input, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
    err = hipMalloc((void**)&d_output, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }

    // Copy input data to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Configure kernel execution
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    int stride = 2;

    // Benchmark non-coalesced kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    nonCoalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N, stride);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print result
    printf("Non-Coalesced Kernel Time (stride=%d): %f ms\n", stride, milliseconds);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
