#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA Kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 1024;
    size_t size = N * sizeof(float);
    hipError_t err;

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors.\n");
        exit(EXIT_FAILURE);
    }

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        exit(EXIT_FAILURE);
    }

    // Copy data from host to device
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the vector addition kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Synchronize to ensure kernel completion
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %s after launching kernel!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the result back to the host
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
